#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <stdint.h>
#include"hmac.cuh"
//#include"aes.cuh"
#define FILELEN 76
#define FILENAME "task3.txt"
__device__ uint8_t passwordG[] = "Fasgdh346ylbne";
#define LEN_PASSWORD 14
#define LENSALT 4
#define CT_LEN (FILELEN - 12)
#define KEYLENGTH 16

__device__ size_t cuda_strlen(uint8_t* str) {
    size_t i = 0;
    while (str[i] != '\0') ++i;
    return i;
}

#define RotWord(value) ((value << 8) | (value >> 24))

__device__
uint8_t SboxG[] = {
        0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76, 
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0, 
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15, 
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75, 
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84, 
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf, 
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8, 
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2, 
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73, 
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb, 
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79, 
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08, 
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a, 
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e, 
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf, 
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
    };

__device__
uint8_t InvSboxG[] = {
        0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
        0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
        0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
        0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
        0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
        0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
        0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
        0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
        0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
        0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
        0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
        0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
        0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
        0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
        0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
        0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d
        };

__device__
uint32_t RconG[] = { 0x00000000, 0x01000000,0x02000000,0x04000000,0x08000000,0x10000000,0x20000000,0x40000000,0x80000000,0x1b000000,0x36000000};

__device__
void InvShiftRow(uint8_t *state) {
    uint8_t temp = state[13];
    state[13] = state[9];
    state[9] = state[5];
    state[5] = state[1];
    state[1] = temp;
    temp = state[14];
    state[14] = state[6];
    state[6] = temp;
    temp = state[10];
    state[10] = state[2];
    state[2] = temp;
    temp = state[15];
    state[15] = state[3];
    state[3] = state[7];
    state[7] = state[11];
    state[11] = temp;
}

__device__
uint32_t SubWord(uint32_t word, uint8_t* Sbox) {
	uint32_t new_word = 0;
	for (int i = 0; i < 4; ++i)
		((uint8_t*)(&new_word))[i] = Sbox[((uint8_t*)(&word))[i]];
	return new_word;
}

__device__
void key_sheldure(uint8_t* key, uint32_t* w, uint8_t* Sbox, uint32_t* Rcon) {
	uint32_t temp;
	int i = 0;
	while (i < 4) {
		((uint8_t*)(w+i))[0] = key[4*i+3];
		((uint8_t*)(w+i))[1] = key[4*i+2];
		((uint8_t*)(w+i))[2] = key[4*i+1];
		((uint8_t*)(w+i))[3] = key[4*i];
		++i;
	}
	while(i < 44) {
		temp = w[i-1];
		if (i % 4 == 0)
			temp = SubWord(RotWord(temp), Sbox) ^ Rcon[i / 4];
		w[i] = w[i - 4] ^ temp;
		++i;
	}
}

__device__
void getWordByIndex(int64_t index, uint8_t* word, uint8_t* password) {
    int64_t i = 0;
    while (index >= 0) {
        int64_t len = i + 1;
        int64_t total = LEN_PASSWORD;

        for (int64_t j = 1; j < len; j++) {
            total *= LEN_PASSWORD;
        }

        if (index < total) {
            int64_t num = index;
            for (int64_t k = len - 1; k >= 0; k--) {
                word[k] = password[num % LEN_PASSWORD];
                num /= LEN_PASSWORD;
            }
            word[len] = '\0';
            return;
        }

        index -= total;
        i++;
    }

    word[0] = '\0';
}

typedef uint8_t state_t[4][4];

__device__
static uint8_t xtime(uint8_t x) {
  return ((x<<1) ^ (((x>>7) & 1) * 0x1b));
}

__device__
static uint8_t Multiply(uint8_t x, uint8_t y) {
  return (((y & 1) * x) ^
       ((y>>1 & 1) * xtime(x)) ^
       ((y>>2 & 1) * xtime(xtime(x))) ^
       ((y>>3 & 1) * xtime(xtime(xtime(x)))) ^
       ((y>>4 & 1) * xtime(xtime(xtime(xtime(x))))));
}

__device__
void InvSubBytes(uint8_t* word, uint8_t* InvSbox) {
	for (int i = 0; i < 16; ++i) word[i] = InvSbox[word[i]];
}

__device__
void invMixColumns(uint8_t* ct) {
    state_t state;
    for (int i = 0; i < 4; ++i)
        for (int j = 0; j < 4; ++j)
            state[i][j] = ct[4*i+j];
    int i;
    uint8_t a, b, c, d;
    for (i = 0; i < 4; ++i) {
        a = state[i][0];
        b = state[i][1];
        c = state[i][2];
        d = state[i][3];

        state[i][0] = Multiply(a, 0x0e) ^ Multiply(b, 0x0b) ^ Multiply(c, 0x0d) ^ Multiply(d, 0x09);
        state[i][1] = Multiply(a, 0x09) ^ Multiply(b, 0x0e) ^ Multiply(c, 0x0b) ^ Multiply(d, 0x0d);
        state[i][2] = Multiply(a, 0x0d) ^ Multiply(b, 0x09) ^ Multiply(c, 0x0e) ^ Multiply(d, 0x0b);
        state[i][3] = Multiply(a, 0x0b) ^ Multiply(b, 0x0d) ^ Multiply(c, 0x09) ^ Multiply(d, 0x0e);
    }
    for (int i = 0; i < 4; ++i)
        for (int j = 0; j < 4; ++j)
            ct[4*i+j] = state[i][j];
}


// Th

/* __device__ uint8_t invMixColumsTable[] = {0x0e, 0x0b, 0x0d, 0x09, 0x09, 0x0e, 0x0b, 0x0d, 0x0d, 0x09, 0x0e, 0x0b, 0x0b, 0x0d, 0x09, 0x0e};

__device__
uint8_t GMul(uint8_t a, uint8_t b) {
    uint8_t p = 0;
    for (int counter = 0; counter < 8; counter++) {
        if ((b & 1) != 0) p ^= a;
        bool hi_bit_set = (a & 0x80) != 0;
        a <<= 1;
        if (hi_bit_set) a ^= 0x1B;
        b >>= 1;
    }
    return p;
}

__device__
void invMixColumns(uint8_t* ct) {
    printf("inv mix col ");
    for(int i = 0; i < 16; ++i) printf("%x ", ct[i]);
    printf("\n");
    uint8_t result[16];
    for (int i = 0; i < 4; ++i)
        for (int j = 0; j < 4; ++j) {
            result[i * 4 + j] = 0;
            for (int k = 0; k < 4; ++k)
                result[i * 4 + j] ^= GMul(invMixColumsTable[i * 4 + k], ct[k * 4 + j]);
        }
    printf("result ");
    for(int i = 0; i < 16; ++i) printf("%x ", result[i]);
    printf("\n");
    memcpy(ct, result, 16);
} */

__device__
void AddRoundKey(uint8_t* text, uint8_t* round_key) {
    for (int i = 0; i < 4; ++i) text[i] ^= round_key[3-i];
    for (int i = 0; i < 4; ++i) text[4+i] ^= round_key[7-i];
    for (int i = 0; i < 4; ++i) text[8+i] ^= round_key[11-i];
    for (int i = 0; i < 4; ++i) text[12+i] ^= round_key[15-i];
}

__device__
void aes_decrypt(uint8_t* ct, uint32_t* w, uint8_t* InvSbox) {
    AddRoundKey(ct, (uint8_t*)(w+40));
    for (int round = 9; round >= 1; --round) {
        InvShiftRow(ct);
        InvSubBytes(ct, InvSbox);
        AddRoundKey(ct, (uint8_t*)(w+(4 * round)));
        invMixColumns(ct);
    }
    InvShiftRow(ct);
    InvSubBytes(ct, InvSbox);
    AddRoundKey(ct, (uint8_t*)w);
}
/*
#define F0(m,l,k) (m&l | ~m&k)
#define F1(m,l,k) (m ^ l ^ k)
#define F2(m,l,k) (m&l | m&k | l&k)
#define F3(m,l,k) (m ^ l ^ k)
#define K0 0x5A827999
#define K1 0x6ED9EBA1
#define K2 0x8F1BBCDC
#define K3 0xCA62C1D6
#define LEFT_ROTATE_32(x,y) (((x) << (y)) | ((x) >> (32-(y)))) */


/* __device__
void sha1(uint8_t* text, u_int64_t len_text, uint8_t* hash) {
    len_text *= 8;
    int padding = len_text % 512 >= 448 ? 512 - (len_text % 512) + 448 : 448 - (len_text % 512);
    int len_padded_text = (len_text + padding + 64) / 8;
    uint8_t* padded_text = (uint8_t*)malloc(len_padded_text);
    memset(padded_text, 0, len_padded_text);
    memcpy(padded_text, text, len_text % 8 == 0 ? len_text / 8 : len_text / 8 + 1);
    padded_text[len_text / 8] |= padding % 8 > 0 ? 1 << (padding % 8 - 1) : 0x80;
    for (int i = 0; i < 64; ++i) padded_text[len_padded_text - 1 - i] = ((uint8_t*)(&len_text))[i];
    uint32_t A = 0x67452301;
    uint32_t B = 0xEFCDAB89;
    uint32_t C = 0x98BADCFE;
    uint32_t D = 0x10325476;
    uint32_t E = 0xC3D2E1F0;
    for(int num_block = 0; num_block < len_padded_text / 64; ++num_block) {
        uint32_t a = A; uint32_t b = B; uint32_t c = C; uint32_t d = D; uint32_t e = E;
        uint32_t W[80];
        for (int i = 0; i < 16; ++i)
            for (int j = 0; j < 4; ++j)
                ((int8_t*)(W+i))[3-j] = padded_text[num_block * 64 + i * 4 + j];
        for (int i = 16; i < 80; ++i) W[i] = LEFT_ROTATE_32(W[i-3]^W[i-8]^W[i-14]^W[i-16], 1);
        for (int i = 0; i < 20; ++i) {
            uint32_t temp = LEFT_ROTATE_32(a,5) + F0(b,c,d) + e + W[i] + K0;
            e = d;
            d = c;
            c = LEFT_ROTATE_32(b,30);
            b = a;
            a = temp;            
        }
        for (int i = 20; i < 40; ++i) {
            uint32_t temp = LEFT_ROTATE_32(a,5) + F1(b,c,d) + e + W[i] + K1;
            e = d;
            d = c;
            c = LEFT_ROTATE_32(b,30);
            b = a;
            a = temp;            
        }
        for (int i = 40; i < 60; ++i) {
            uint32_t temp = LEFT_ROTATE_32(a,5) + F2(b,c,d) + e + W[i] + K2;
            e = d;
            d = c;
            c = LEFT_ROTATE_32(b,30);
            b = a;
            a = temp;            
        }
        for (int i = 60; i < 80; ++i) {
            uint32_t temp = LEFT_ROTATE_32(a,5) + F3(b,c,d) + e + W[i] + K3;
            e = d;
            d = c;
            c = LEFT_ROTATE_32(b,30);
            b = a;
            a = temp;            
        }
        A += a; B+=b, C+=c, D+=d; E+=e;
    }
    for (int i = 0; i < 4; ++i) hash[i] = ((uint8_t*)(&A))[3-i];
    for (int i = 0; i < 4; ++i) hash[4+i] = ((uint8_t*)(&B))[3-i];
    for (int i = 0; i < 4; ++i) hash[8+i] = ((uint8_t*)(&C))[3-i];
    for (int i = 0; i < 4; ++i) hash[12+i] = ((uint8_t*)(&D))[3-i];
    for (int i = 0; i < 4; ++i) hash[16+i] = ((uint8_t*)(&E))[3-i];
    free(padded_text);
} */

/* __device__
void HMAC_SHA1(uint8_t* text, size_t len_text, uint8_t* key, size_t len_key, uint8_t* result) {
    uint8_t block_sized_key[64];
    memset(block_sized_key, 0, 64);
    if (len_key > 64) {
        sha1(key, len_key, block_sized_key);
    }
    else memcpy(block_sized_key, key, len_key);
    uint8_t o_key_pad[64];
    uint8_t i_key_pad[64];
    for (int i = 0; i < 64; ++i) {
        o_key_pad[i] = block_sized_key[i] ^ 0x5c;
        i_key_pad[i] = block_sized_key[i] ^ 0x36;
    }
    uint8_t* buf = (uint8_t*)malloc(64+len_text);
    memcpy(buf, i_key_pad, 64);
    memcpy(buf+64, text, len_text);
    sha1(buf, 64+len_text, result);
    free(buf);
    uint8_t buf2[84];
    memcpy(buf2, o_key_pad, 64);
    memcpy(buf2 + 64, result, 20);
    sha1(buf2, 84, result);
} */

__device__
void PBKDF2_HMAC_SHA1(uint8_t* passwd, uint8_t* salt, int counter, uint8_t* key) {
    size_t len_digest = 20;
    memset(key, 0, 20);
    uint8_t U[20] = {0};
    size_t len_passwd = cuda_strlen(passwd);
    uint8_t buf[8];
    memcpy(buf, salt, 4);
    buf[7] = (uint8_t)0x01;
    memset(buf + 4, 0, 3);
    //HMAC_SHA1(buf, 8, passwd, len_passwd, U);
    hmac_sha1(passwd, len_passwd, buf, 8, U, &len_digest);
    for(int i = 0; i < 20; ++i) key[i] ^= U[i];
    for (int i = 1; i < counter; ++i) {
        //HMAC_SHA1(U, 20, passwd, len_passwd, U);
        hmac_sha1(passwd,len_passwd,U,20,U,&len_digest);
        for(int i = 0; i < 20; ++i) key[i] ^= U[i];
    }
}

__global__
void decrypt(uint8_t* ct, uint8_t* salt, int counter, int pt_size, int64_t start_index) { 
    __shared__ uint8_t passwordS[LEN_PASSWORD];
    __shared__ uint8_t SboxS[256];
    __shared__ uint8_t InvSboxS[256];
    __shared__ uint32_t RconS[11];

    if (threadIdx.x < LEN_PASSWORD)
        passwordS[threadIdx.x] = passwordG[threadIdx.x];
    if (threadIdx.x < 256) {
        SboxS[threadIdx.x] = SboxG[threadIdx.x];
        InvSboxS[threadIdx.x] = InvSboxG[threadIdx.x];
    }
    if (threadIdx.x < 11)
        RconS[threadIdx.x] = RconG[threadIdx.x]; 

    __syncthreads();

    uint8_t text[CT_LEN];
    memcpy(text, ct, CT_LEN);

    int64_t gid = ((((int64_t)blockIdx.x + ((int64_t)blockIdx.y * (int64_t)gridDim.x) + ((int64_t)blockIdx.z * ((int64_t)gridDim.x * (int64_t)gridDim.y))) * ((int64_t)blockDim.x * (int64_t)blockDim.y * (int64_t)blockDim.z)) + ((int64_t)threadIdx.z * ((int64_t)blockDim.x * (int64_t)blockDim.y)) + ((int64_t)threadIdx.y * (int64_t)blockDim.x) + (int64_t)threadIdx.x);
    int64_t index = gid + start_index;
    
    
    uint8_t key[20];
    uint8_t word[LEN_PASSWORD];
    getWordByIndex(index, word, passwordS);
    PBKDF2_HMAC_SHA1(word,salt,counter, key);
    uint32_t key_sheld[44];
    key_sheldure(key, key_sheld, SboxS, RconS);
    //AES_ctx ctx;
    //AES_init_ctx(&ctx, key, SboxS, RconS);
    for (int i = 0; i < CT_LEN; i += 16)
        aes_decrypt(text + i, key_sheld, InvSboxS);
        //AES_ECB_decrypt(&ctx, text+i, SboxS, InvSboxS);
    int acceptable = 0;
    for (int i = 0; i < pt_size; ++i)
        if ((text[i] == '\n') || (text[i] >= ' ') && (text[i] <= '~'))
            ++acceptable;
    if (acceptable == pt_size) {
        text[pt_size] = (uint8_t)0;
        printf("%s %s\n", word, text);
    }
    if (index % 1475789056 == 0)
        printf("%li\n", index);
}

int main() {
    FILE* fp = fopen(FILENAME, "rb");
    if (fp == NULL) {
        printf("Can't open file\n");
        return -1;
    }
    uint8_t buf[FILELEN];
    for (int i = 0; i < FILELEN; ++i) buf[i] = (uint8_t)getc(fp);
    fclose(fp);
    uint8_t* salt;
    uint8_t* ct;
    hipMalloc(&salt, 4);
    hipMalloc(&ct, CT_LEN);
    uint32_t counter;
    uint32_t pt_size;
    for (int i = 0; i < 4; ++i) {
        ((uint8_t*)(&counter))[3-i] = buf[4+i];
        ((uint8_t*)(&pt_size))[3-i] = buf[8+i]; 
    }
    printf("%u %u\n", counter, pt_size);
    hipMemcpy(salt, buf, 4, hipMemcpyHostToDevice);
    hipMemcpy(ct, buf+12, CT_LEN, hipMemcpyHostToDevice);
    int64_t tries = 854769755812155;
    for (int64_t i = 67886296576; i < tries; i+=((int64_t)1<<30)*(int64_t)256) {
        decrypt<<<(1<<30),256>>>(ct, salt, counter, pt_size, i);
        hipDeviceSynchronize();
        printf("end of set of threads %li\n", i);
    }
    /* decrypt<<<1,1>>>(ct, salt, counter, pt_size, 0);
    hipDeviceSynchronize(); */
}